#include "head.h"

double *h_t;
double *d_t;
double *h_V;
double *d_V;
double *d_dV2;
double *d_Vnew;
double *d_it;

double *h_m;
double *d_m;
double *h_h;
double *d_h;
double *h_jj;
double *d_jj;
double *h_d;
double *d_d;
double *h_f;
double *d_f;
double *h_X;
double *d_X;
double *h_cai;
double *d_cai;

double *h_it;

double *dcai;

/* FVM variable*/
int *h_stim_i;
int *h_adj_triangle; // save the index of  three neighbor triangles
double *h_weight; //weighting factor
double *h_D_CF1;
double *h_D_CF2;
double *h_D_CF3;
double *h_matrix_G; //for Least - Square method in Section 9.3
double *h_gC; //geometric interpolation factor related to the position of the element face f with respect to the nodes C and F.
double *h_area; // each triangle's h_area
double *h_Sf;//surface vector
double *h_unit_CF;
double *h_Tf; // component Tf is normal to Sf
double *h_aF;

int *d_stim_i;
int *d_adj_triangle; // save the index of  three neighbor triangles
double *d_weight; //weighting factor
double *d_D_CF1;
double *d_D_CF2;
double *d_D_CF3;
double *d_matrix_G; //for Least - Square method in Section 9.3
double *d_gC; //geometric interpolation factor related to the position of the element face f with respect to the nodes C and F.
double *d_area; // each triangle's h_area
double *d_Sf;//surface vector
double *d_unit_CF;
double *d_Tf; // component Tf is normal to Sf
double *d_aF;

double *d_gradientU_C;

void Allocate(){
	hipError_t Error;
	size_t size = nt*sizeof(double);

	h_t = (double*)malloc(size);
	Error = hipMalloc((void**)&d_t, size);
	printf("CUDA error = %s\n",hipGetErrorString(Error));

	h_V = (double*)malloc(size);
	hipMalloc((void**)&d_V, size);
	hipMalloc((void**)&d_dV2, size);
	hipMalloc((void**)&d_Vnew, size);

	hipMalloc((void**)&d_it, size);

	h_m = (double*)malloc(size);
	hipMalloc((void**)&d_m, size);
	h_h = (double*)malloc(size);
        hipMalloc((void**)&d_h, size);
	h_jj = (double*)malloc(size);
        hipMalloc((void**)&d_jj, size);
	h_d = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
	h_f = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
	h_X = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
	h_cai = (double*)malloc(size);
        hipMalloc((void**)&d_cai, size);

	h_it = (double*)malloc(size);

	hipMalloc((void**)&dcai, size);
	
	h_stim_i = (int *)malloc(nt*sizeof(int));
	h_adj_triangle = (int *)malloc(nt*3*sizeof(int));
	h_weight = (double *)malloc(nt*3*sizeof(double));
	h_D_CF1 = (double *)malloc(nt*2*sizeof(double));
	h_D_CF2 = (double *)malloc(nt*2*sizeof(double));
	h_D_CF3 = (double *)malloc(nt*2*sizeof(double));
	h_matrix_G = (double *)malloc(nt*2*2*sizeof(double));
	h_gC = (double *)malloc(nt*3*sizeof(double));
	h_area = (double *)malloc(nt*sizeof(double));
	h_Sf = (double *)malloc(nt*3*2*sizeof(double));
	h_unit_CF = (double *)malloc(nt*3*2*sizeof(double));
	h_Tf = (double *)malloc(nt*3*2*sizeof(double));
	h_aF = (double *)malloc(nt*3*sizeof(double));
	
	hipMalloc((void**)&d_stim_i, nt*sizeof(int));
	hipMalloc((void**)&d_adj_triangle, nt*3*sizeof(int));
	hipMalloc((void**)&d_weight, nt*3*sizeof(double));
	hipMalloc((void**)&d_D_CF1, nt*2*sizeof(double));
	hipMalloc((void**)&d_D_CF2, nt*2*sizeof(double));
	hipMalloc((void**)&d_D_CF3, nt*2*sizeof(double));
	hipMalloc((void**)&d_matrix_G, nt*2*2*sizeof(double));
	hipMalloc((void**)&d_gC, nt*3*sizeof(double));
	hipMalloc((void**)&d_area, nt*sizeof(double));
	hipMalloc((void**)&d_Sf, nt*3*2*sizeof(double));
	hipMalloc((void**)&d_unit_CF, nt*3*2*sizeof(double));
	hipMalloc((void**)&d_Tf, nt*3*2*sizeof(double));
	hipMalloc((void**)&d_aF, nt*3*sizeof(double));
	
	hipMalloc((void**)&d_gradientU_C, nt*2*sizeof(double));	
}

void Free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_it);

	hipFree(d_t);hipFree(d_V);hipFree(d_dV2);hipFree(d_Vnew);hipFree(d_it);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_f);hipFree(d_X);hipFree(d_cai);

	hipFree(dcai);

	free(h_stim_i);free(h_adj_triangle);free(h_weight);
	free(h_D_CF1);free(h_D_CF2);free(h_D_CF3);
	free(h_matrix_G);free(h_gC);free(h_area);
	free(h_Sf);free(h_unit_CF);free(h_Tf);free(h_aF);
	
	hipFree(d_stim_i);hipFree(d_adj_triangle);hipFree(d_weight);
	hipFree(d_D_CF1);hipFree(d_D_CF2);hipFree(d_D_CF3);
	hipFree(d_matrix_G);hipFree(d_gC);hipFree(d_area);
	hipFree(d_Sf);hipFree(d_unit_CF);hipFree(d_Tf);hipFree(d_aF);
	hipFree(d_gradientU_C);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nt*sizeof(double);

	Error = hipMemcpy(d_t, h_t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_stim_i, h_stim_i, nt*sizeof(int), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_adj_triangle, h_adj_triangle, nt*3*sizeof(int), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_weight, h_weight, nt*3*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_D_CF1, h_D_CF1, nt*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_D_CF2, h_D_CF2, nt*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_D_CF3, h_D_CF3, nt*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_matrix_G, h_matrix_G, nt*2*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_gC, h_gC, nt*3*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_area, h_area, nt*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_Sf, h_Sf, nt*3*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_unit_CF, h_unit_CF, nt*3*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_Tf, h_Tf, nt*3*2*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_aF, h_aF, nt*3*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_stim_i->d_stim_i) = %s\n",hipGetErrorString(Error));

}

void Send_V(){
        hipError_t Error;
        size_t size;
        size = nt*sizeof(double);

        Error = hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
}
